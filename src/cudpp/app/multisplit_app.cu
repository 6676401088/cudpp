#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// -------------------------------------------------------------
// This source code is distributed under the terms of license.txt
// in the root directory of this source distribution.
// -------------------------------------------------------------

/**
 * @file
 * multisplit_app.cu
 *
 * @brief CUDPP application-level multisplit routines
 */

/** @addtogroup cudpp_app
 * @{
 */

/** @name MultiSplit Functions
 * @{
 */
#include <hipcub/hipcub.hpp>
#include "cuda_util.h"
#include "cudpp.h"
#include "cudpp_util.h"
#include "limits.h"
#include "kernel/multisplit_kernel.cuh"


//===============================================
// Global
//===============================================
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
typedef unsigned long long int uint64;
//===============================================
// Definitions:
//===============================================
#define NUM_WARPS 8
#define LOG_WARPS 3 // = ceil(log2(NUM_WARPS))
#define SMEM_BUCK_SIZE (1536/(NUM_BUCKETS*NUM_WARPS))
#define PACK_DEPTH 4
#define PACK_PRE 8
#define PACK_POST 4

#define BLOCKSORT_SIZE 1024
#define DEPTH 8

/** @brief Performs merge sort utilizing 3 stages:
 * (1) Blocksort, (2) simple merge and (3) multi merge
 *
 *
 * @param[in,out] pkeys Keys to be sorted.
 * @param[in,out] pvals Associated values to be sorted
 * @param[in] numElements Number of elements in the sort.
 * @param[in] plan Configuration information for mergesort.
 **/
void runMultiSplit(unsigned int *d_inp, uint numElements, uint numBuckets, const CUDPPMultiSplitPlan *plan) {
  unsigned int nB = ceil(numElements / (NUM_WARPS * 32));
  unsigned int NT = NUM_WARPS * 32;
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;

  if (numBuckets == 1)
    return;

  if (numBuckets == 2) {
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo, plan->m_d_histo, numBuckets * nB * NUM_WARPS * PACK_DEPTH);
  } else if (numBuckets <= 32) {
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo, plan->m_d_histo, numBuckets * nB * PACK_DEPTH);
  } else if (numBuckets > 96) {
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, plan->m_d_mask, plan->m_d_out, d_inp, plan->m_d_fin,
        numElements, 0, plan->m_logBuckets);
  } else if (numBuckets <= 96){
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo, plan->m_d_histo, numBuckets * nB);
  } else {
    printf("Bad number of buckets: %u\n", numBuckets);
    return;
  }
  g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);

  if (numBuckets == 2) {
    histogram_warp<NUM_WARPS, 2, 1, PACK_PRE> <<<nB / PACK_PRE, NT>>>(d_inp,
        plan->m_d_histo, numElements);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo, plan->m_d_histo, 2 * nB * NUM_WARPS * PACK_DEPTH);
    split_WMS<NUM_WARPS, 2, 1, PACK_POST><<<nB/PACK_POST, NT>>>(d_inp, plan->m_d_histo, plan->m_d_fin, numElements);
  } else if (numBuckets <= 32) {
    histogram_block<<<nB / PACK_PRE, NT,
        NUM_WARPS * numBuckets * DEPTH * sizeof(uint)>>>(d_inp, plan->m_d_histo,
        numElements, numBuckets, NUM_WARPS, PACK_PRE);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo, plan->m_d_histo, numBuckets * nB * PACK_DEPTH);
    split_BMS<<<nB / PACK_POST, NT,
        (2 * numBuckets * PACK_POST + 32 * NUM_WARPS * PACK_POST
            + numBuckets * NUM_WARPS * PACK_POST) * sizeof(uint)>>>(d_inp, plan->m_d_histo, plan->m_d_fin,
        numElements, numBuckets, NUM_WARPS, PACK_POST);
  } else if (numBuckets > 96) {
    markBins_general<<<nB, NT>>>(plan->m_d_mask, d_inp, numElements, numBuckets);
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, plan->m_d_mask,
        plan->m_d_out, d_inp, plan->m_d_fin, numElements, 0,
        int(ceil(log2(float(numBuckets)))));
  } else if (numBuckets <= 96) {
    switch(numBuckets){
      case 33:
        histogramBallot_Mode13_large<NUM_WARPS, 33, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 33 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 33> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 34:
        histogramBallot_Mode13_large<NUM_WARPS, 34, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 34 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 34> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 35:
        histogramBallot_Mode13_large<NUM_WARPS, 35, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 35 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 35> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 36:
        histogramBallot_Mode13_large<NUM_WARPS, 36, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 36 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 36> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 37:
        histogramBallot_Mode13_large<NUM_WARPS, 37, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 37 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 37> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 38:
        histogramBallot_Mode13_large<NUM_WARPS, 38, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 38 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 38> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 39:
        histogramBallot_Mode13_large<NUM_WARPS, 39, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 39 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 39> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 40:
        histogramBallot_Mode13_large<NUM_WARPS, 40, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 40 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 40> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 41:
        histogramBallot_Mode13_large<NUM_WARPS, 41, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 41 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 41> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 42:
        histogramBallot_Mode13_large<NUM_WARPS, 42, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 42 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 42> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 43:
        histogramBallot_Mode13_large<NUM_WARPS, 43, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 43 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 43> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 44:
        histogramBallot_Mode13_large<NUM_WARPS, 44, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 44 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 44> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 45:
        histogramBallot_Mode13_large<NUM_WARPS, 45, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 45 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 45> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 46:
        histogramBallot_Mode13_large<NUM_WARPS, 46, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 46 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 46> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 47:
        histogramBallot_Mode13_large<NUM_WARPS, 47, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 47 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 47> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 48:
        histogramBallot_Mode13_large<NUM_WARPS, 48, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 48 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 48> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 49:
        histogramBallot_Mode13_large<NUM_WARPS, 49, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 49 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 49> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 50:
        histogramBallot_Mode13_large<NUM_WARPS, 50, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 50 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 50> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 51:
        histogramBallot_Mode13_large<NUM_WARPS, 51, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 51 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 51> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 52:
        histogramBallot_Mode13_large<NUM_WARPS, 52, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 52 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 52> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 53:
        histogramBallot_Mode13_large<NUM_WARPS, 53, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 53 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 53> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 54:
        histogramBallot_Mode13_large<NUM_WARPS, 54, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 54 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 54> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 55:
        histogramBallot_Mode13_large<NUM_WARPS, 55, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 55 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 55> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 56:
        histogramBallot_Mode13_large<NUM_WARPS, 56, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 56 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 56> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 57:
        histogramBallot_Mode13_large<NUM_WARPS, 57, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 57 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 57> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 58:
        histogramBallot_Mode13_large<NUM_WARPS, 58, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 58 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 58> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 59:
        histogramBallot_Mode13_large<NUM_WARPS, 59, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 59 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 59> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 60:
        histogramBallot_Mode13_large<NUM_WARPS, 60, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 60 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 60> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 61:
        histogramBallot_Mode13_large<NUM_WARPS, 61, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 61 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 61> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 62:
        histogramBallot_Mode13_large<NUM_WARPS, 62, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 62 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 62> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 63:
        histogramBallot_Mode13_large<NUM_WARPS, 63, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 63 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 63> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 64:
        histogramBallot_Mode13_large<NUM_WARPS, 64, 6, LOG_WARPS> <<<
          nB, NT>>>(d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, plan->m_d_histo,
          plan->m_d_histo, 64 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 64> <<<nB, NT>>>(d_inp, plan->m_d_histo,
          plan->m_d_fin, numElements);
      break;
      case 65:
        histogramBallot_Mode13_large<NUM_WARPS, 65, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 65 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 65> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 66:
        histogramBallot_Mode13_large<NUM_WARPS, 66, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 66 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 66> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 67:
        histogramBallot_Mode13_large<NUM_WARPS, 67, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 67 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 67> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 68:
        histogramBallot_Mode13_large<NUM_WARPS, 68, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 68 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 68> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 69:
        histogramBallot_Mode13_large<NUM_WARPS, 69, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 69 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 69> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 70:
        histogramBallot_Mode13_large<NUM_WARPS, 70, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 70 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 70> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 71:
        histogramBallot_Mode13_large<NUM_WARPS, 71, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 71 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 71> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 72:
        histogramBallot_Mode13_large<NUM_WARPS, 72, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 72 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 72> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 73:
        histogramBallot_Mode13_large<NUM_WARPS, 73, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 73 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 73> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 74:
        histogramBallot_Mode13_large<NUM_WARPS, 74, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 74 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 74> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 75:
        histogramBallot_Mode13_large<NUM_WARPS, 75, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 75 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 75> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 76:
        histogramBallot_Mode13_large<NUM_WARPS, 76, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 76 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 76> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 77:
        histogramBallot_Mode13_large<NUM_WARPS, 77, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 77 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 77> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 78:
        histogramBallot_Mode13_large<NUM_WARPS, 78, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 78 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 78> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 79:
        histogramBallot_Mode13_large<NUM_WARPS, 79, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 79 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 79> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 80:
        histogramBallot_Mode13_large<NUM_WARPS, 80, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 80 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 80> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 81:
        histogramBallot_Mode13_large<NUM_WARPS, 81, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 81 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 81> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 82:
        histogramBallot_Mode13_large<NUM_WARPS, 82, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 82 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 82> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 83:
        histogramBallot_Mode13_large<NUM_WARPS, 83, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 83 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 83> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 84:
        histogramBallot_Mode13_large<NUM_WARPS, 84, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 84 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 84> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 85:
        histogramBallot_Mode13_large<NUM_WARPS, 85, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 85 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 85> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 86:
        histogramBallot_Mode13_large<NUM_WARPS, 86, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 86 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 86> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 87:
        histogramBallot_Mode13_large<NUM_WARPS, 87, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 87 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 87> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 88:
        histogramBallot_Mode13_large<NUM_WARPS, 88, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 88 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 88> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 89:
        histogramBallot_Mode13_large<NUM_WARPS, 89, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 89 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 89> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 90:
        histogramBallot_Mode13_large<NUM_WARPS, 90, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 90 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 90> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 91:
        histogramBallot_Mode13_large<NUM_WARPS, 91, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 91 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 91> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 92:
        histogramBallot_Mode13_large<NUM_WARPS, 92, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 92 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 92> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 93:
        histogramBallot_Mode13_large<NUM_WARPS, 93, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 93 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 93> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 94:
        histogramBallot_Mode13_large<NUM_WARPS, 94, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 94 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 94> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 95:
        histogramBallot_Mode13_large<NUM_WARPS, 95, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 95 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 95> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      case 96:
        histogramBallot_Mode13_large<NUM_WARPS, 96, 7, LOG_WARPS> <<<nB, NT>>>(
            d_inp, plan->m_d_histo, numElements);
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
            plan->m_d_histo, plan->m_d_histo, 96 * nB);
        splitBallot_Mode13_large<NUM_WARPS, 96> <<<nB, NT>>>(d_inp,
            plan->m_d_histo, plan->m_d_fin, numElements);
        break;
      default:
        break;
    }
  }

  CUDA_SAFE_CALL(hipMemcpy(d_inp, plan->m_d_fin, numElements*sizeof(unsigned int), hipMemcpyDeviceToDevice));

  if(d_temp_storage)
    CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
}

#ifdef __cplusplus
extern "C"
{
#endif

/**
 * @brief From the programmer-specified sort configuration,
 *        creates internal memory for performing the sort.
 *
 * @param[in] plan Pointer to CUDPPMergeSortPlan object
 **/
void allocMultiSplitStorage(CUDPPMultiSplitPlan *plan)
{
  unsigned int nB = ceil(plan->m_numElements / (NUM_WARPS * 32));

  printf("NUM ALLOCATED BYTES: %u\n",
      (plan->m_numElements + 1)
          * sizeof(unsigned int) + plan->m_numElements * sizeof(unsigned int)+
          sizeof(unsigned int) * plan->m_numBuckets * nB * NUM_WARPS * 2 * PACK_DEPTH + plan->m_numElements*sizeof(unsigned int));

  if (plan->m_numBuckets > 96) {
    CUDA_SAFE_CALL(hipMalloc((void**) &plan->m_d_mask, (plan->m_numElements+1)*sizeof(unsigned int)));  // mask verctor, +1 added only for the near-far implementation
    CUDA_SAFE_CALL(hipMalloc((void**) &plan->m_d_out, plan->m_numElements*sizeof(unsigned int))); // gpu output
  }
  CUDA_SAFE_CALL(
      hipMalloc((void**) &plan->m_d_histo, sizeof(unsigned int) * plan->m_numBuckets * nB * NUM_WARPS * 2 * PACK_DEPTH)); //
  CUDA_SAFE_CALL(hipMalloc((void**) &plan->m_d_fin, plan->m_numElements*sizeof(unsigned int))); // final masks (used for reduced bit method, etc.)

  if (plan->m_numBuckets > 96) {
    CUDA_SAFE_CALL(hipMemset(plan->m_d_mask, 0, sizeof(unsigned int)*(plan->m_numElements+1)));
    CUDA_SAFE_CALL(hipMemset(plan->m_d_out, 0, sizeof(unsigned int)*plan->m_numElements));
  }
  CUDA_SAFE_CALL(hipMemset(plan->m_d_histo, 0, sizeof(unsigned int) * plan->m_numBuckets * nB * NUM_WARPS * 2
      * PACK_DEPTH));
  CUDA_SAFE_CALL(hipMemset(plan->m_d_fin, 0, sizeof(unsigned int)*plan->m_numElements));
}

/** @brief Deallocates intermediate memory from allocRadixSortStorage.
 *
 *
 * @param[in] plan Pointer to CUDPPMergeSortPlan object
 **/

void freeMultiSplitStorage(CUDPPMultiSplitPlan* plan)
{
  if (plan->m_numBuckets > 96) {
    hipFree (plan->m_d_mask);
    hipFree (plan->m_d_out);
  }
  hipFree(plan->m_d_histo);
  hipFree(plan->m_d_fin);
}

/** @brief Dispatch function to perform a sort on an array with
 * a specified configuration.
 *
 * This is the dispatch routine which calls mergeSort...() with
 * appropriate template parameters and arguments as specified by
 * the plan.
 * Currently only sorts keys of type int, unsigned int, and float.
 * @param[in,out] keys Keys to be sorted.
 * @param[in,out] values Associated values to be sorted (through keys).
 * @param[in] numElements Number of elements in the sort.
 * @param[in] plan Configuration information for mergeSort.
 **/

void cudppMultiSplitDispatch(unsigned int *elements,
                            size_t numElements,
                            size_t numBuckets,
                            const CUDPPMultiSplitPlan *plan)
{
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

  runMultiSplit(elements, numElements, numBuckets, plan);
}

#ifdef __cplusplus
}
#endif

/** @} */ // end mergesort functions
/** @} */ // end cudpp_app
