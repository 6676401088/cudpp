#include "hip/hip_runtime.h"
/*
* Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32

#define LESS_THAN -1
#define EQUALS 0
#define GREATER_THAN 1

__device__ int compare_to(unsigned int str_ptr1, unsigned int str_ptr2, char *str, unsigned int str_length)
{
   unsigned int index1 = str_ptr1;
   unsigned int index2 = str_ptr2;

   for (unsigned int count = 0; count < str_length; count++) {
      if (str[index1] < str[index2]) {
         return LESS_THAN;
      }

      if (str[index1] > str[index2]) {
         return GREATER_THAN;
      }  

      index1++;
      index2++;

      if (index1 == str_length) {
         index1 = 0;
      }
      if (index2 == str_length) {
         index2 = 0;
      }
   }

   return EQUALS;
}


////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(unsigned int *data, unsigned int left, unsigned int right, char *str, unsigned int str_length )
{
  for( unsigned int i = left ; i <= right ; ++i )
  {
    unsigned int min_val = data[i];
    unsigned int min_idx = i;

    // Find the smallest value in the range [left, right].
    for( int j = i+1 ; j <= right ; ++j )
    {
      unsigned int val_j = data[j];
      if(compare_to(val_j, min_val, str, str_length ) == LESS_THAN)
      {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if( i != min_idx )
    {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, unsigned int left, unsigned int right,
                                    unsigned int depth,  char *str,  unsigned int str_length)
{
  // If we're too deep or there are few elements left, we use an insertion sort...
  if( depth >= MAX_DEPTH || right-left <= INSERTION_SORT )
  {
    selection_sort( data, left, right, str, str_length );
    return;
  }

  unsigned int *lptr = data+left;
  unsigned int *rptr = data+right;
  unsigned int  pivot = data[(left+right)/2];

  // Do the partitioning.
  while(lptr <= rptr)
  {
    // Find the next left- and right-hand values to swap
    int lval = *lptr; 
    int rval = *rptr;

    // Move the left pointer as long as the pointed element is smaller than the pivot.
    while( compare_to(lval, pivot , str, str_length) == LESS_THAN)
    {
      lptr++;
      lval = *lptr;
    }

    // Move the right pointer as long as the pointed element is larger than the pivot.
    while(  compare_to(rval, pivot, str, str_length) == GREATER_THAN)
    {
      rptr--;
      rval = *rptr;
    }

    // If the swap points are valid, do the swap!
    if(lptr <= rptr)
    {
      *lptr++ = rval;
      *rptr-- = lval;
    }
  }

  // Now the recursive part
  unsigned int nright = rptr - data;
  unsigned int nleft  = lptr - data;

  // Launch a new block to sort the left part.
  if(left < (rptr-data)) 
  {
    hipStream_t s;
    hipStreamCreateWithFlags( &s, hipStreamNonBlocking );
    cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1, str, str_length);
    hipStreamDestroy( s );
  }

  // Launch a new block to sort the right part.
  if((lptr-data) < right) 
  {
    hipStream_t s1;
    hipStreamCreateWithFlags( &s1, hipStreamNonBlocking );
    cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1, str, str_length);
    hipStreamDestroy( s1 );
  }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems, char *str)
{
  // Prepare CDP for the max depth 'MAX_DEPTH'.
  checkCudaErrors( hipDeviceSetLimit( cudaLimitDevRuntimeSyncDepth, MAX_DEPTH ) );

  // Launch on device
  unsigned int left = 0;
  unsigned int right = nitems-1;
  std::cout << "Launching kernel on the GPU" << std::endl;
  cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0, str, nitems);
  checkCudaErrors(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems, char *str)
{
  // Fixed seed for illustration
  srand(2047);

  // Fill dst with random values
  for (unsigned i = 0 ; i < nitems ; i++) {
    //dst[i] = rand() % nitems ;
    dst[i] = i;
    str[i] = (char) (nitems - i - 1);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(unsigned int n, unsigned int *results_d )
{
  unsigned int *results_h = new unsigned[n];
  checkCudaErrors( hipMemcpy( results_h, results_d, n*sizeof(unsigned), hipMemcpyDeviceToHost ));
  /*for( unsigned int i = 1 ; i < n ; ++i )
    if( results_h[i-1] > results_h[i] )
    {
      std::cout << "Invalid item[" << i-1 << "]: " << results_h[i-1] << " greater than " << results_h[i] << std::endl;
      exit(EXIT_FAILURE);
    }
  std::cout << "OK" << std::endl;*/

  for( unsigned int i = 0; i < n ; ++i )
    std::cout << results_h[i] << " ";  

  delete[] results_h;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  unsigned int num_items = 128;
  bool verbose = false;

  if (checkCmdLineFlag( argc, (const char **)argv, "help" ) ||
	  checkCmdLineFlag( argc, (const char **)argv, "h" ))
  {
      std::cerr << "Usage: " << argv[0] << " num_items=<num_items>\twhere num_items is the number of items to sort" << std::endl;
      exit(EXIT_SUCCESS);
  }

  if (checkCmdLineFlag( argc, (const char **)argv, "v"))
  {
      verbose = true;
  }
  if (checkCmdLineFlag( argc, (const char **)argv, "num_items"))
  {
      num_items = getCmdLineArgumentInt( argc, (const char **)argv, "num_items");
      if( num_items < 1 )
      {
        std::cerr << "ERROR: num_items has to be greater than 1" << std::endl;
        exit(EXIT_FAILURE);
      }
  }

  // Get device properties
  int device_count = 0, device = -1;
  checkCudaErrors( hipGetDeviceCount( &device_count ) );
  for( int i = 0 ; i < device_count ; ++i )
  {
    hipDeviceProp_t properties;
    checkCudaErrors( hipGetDeviceProperties( &properties, i ) );
    if( properties.major > 3 || ( properties.major == 3 && properties.minor >= 5 ) )
    {
      device = i;
      std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
      break;
    }
    std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
  }
  if( device == -1 )
  {
    std::cerr << "cdpSimpleQuicksort requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
    exit(EXIT_SUCCESS);
  }
  hipSetDevice(device);

  // Create input data
  unsigned int *h_data = 0;
  unsigned int *d_data = 0;
  char str[num_items];
  char *d_str;

  // Allocate CPU memory and initialize data.
  std::cout << "Initializing data:" << std::endl;
  h_data =(unsigned int *)malloc( num_items*sizeof(unsigned int));
  initialize_data(h_data, num_items, str);
  if( verbose )
  {
    for(int i=0 ; i<num_items ; i++)
      std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;
  }
  
  // Allocate GPU memory.
  checkCudaErrors(hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **)&d_str, num_items * sizeof(char)));
  checkCudaErrors(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_str, str, num_items * sizeof(char), hipMemcpyHostToDevice));

  // Execute
  std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
  run_qsort(d_data, num_items, d_str);
  
  // Check result
  std::cout << "Validating results: ";
  check_results(num_items, d_data);

  free(h_data);
  checkCudaErrors( hipFree(d_data));
  checkCudaErrors( hipFree(d_str));
hipDeviceReset();
  exit( EXIT_SUCCESS );
}

