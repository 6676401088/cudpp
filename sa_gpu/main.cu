#include "skew.h"
#include <fstream>

using namespace std;
using namespace SA;
//using namespace mgpu;
typedef unsigned int uint;

int main(int argc, char** argv)
{
  //ContextPtr context = CreateCudaDevice(argc, argv, true);
GpuTimer Timer;
  if (argc!=2) cout << "Usage: ./exefile InputFile" << endl;
  else{
    vector<string> line_text(1000000); //this value is as large as 4294967295 to deal with 4GB data
    ifstream infile;
    infile.open(argv[1]);
    int idx = 0;
    while(!infile.eof())
    {
        getline(infile, line_text[idx++]);
    }
    idx--;
    int str_length = 0;
    for (int i = 0; i < idx; ++i)
    {
        str_length += line_text[i].length();
    }
    char* str = new char[str_length+4];
    unsigned int addr = 0;
    for (int i = 0; i < idx; ++i)
    {
        memcpy(str+addr, line_text[i].c_str(), sizeof(char)*line_text[i].length());
        addr+=line_text[i].length();
    }
    str[str_length] = '$';
    str[str_length+1] = '$';
    str[str_length+2] = '$';


    uint *keys_sa = new uint[str_length+1];
    uint *str_value= new uint [str_length+3];
    uint* d_str;
    uint* d_keys_sa;
/* size_t free_byte ;
 size_t total_byte ;
 hipError_t  cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
 if ( hipSuccess != cuda_status ){
   printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
   exit(EXIT_FAILURE);
 }
cout << "free mem=" << free_byte << endl;
cout << "total mem=" << total_byte <<endl; */ 
    CUDA_SAFE_CALL(hipMalloc((void**)&d_str, (str_length+3)*sizeof(uint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_keys_sa, (str_length+1)*sizeof(uint)));
    //unsigned char *str_value= new unsigned char [str_length+3];
    for (int i=0;i<str_length;i++) str_value[i]=(uint) str[i];       
    for(int i=str_length;i<str_length+3;i++) str_value[i]=0;

    CUDA_SAFE_CALL(hipMemcpy(d_str, str_value, (str_length+3)*sizeof(uint), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(str_value, d_str, (str_length+3)*sizeof(uint), hipMemcpyDeviceToHost));
/*ofstream myfile;
myfile.open("checkResult.txt");*/
  /* for (int i = 0; i < str_length+1; ++i)
    {
        cout << str[i];
    }
    cout << endl;
for (int i = 0; i < str_length+3; ++i)
    {
        cout << str_value[i] << " ";
    }
    cout << endl;*/
/*cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
 if ( hipSuccess != cuda_status ){
   printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
   exit(EXIT_FAILURE);
 }
cout << "free mem=" << free_byte << endl;
cout << "total mem=" << total_byte <<endl; */
Timer.Start();
    runComputeSA(d_str, d_keys_sa, str_length);
Timer.Stop();

    CUDA_SAFE_CALL(hipMemcpy(keys_sa, d_keys_sa, (str_length+1)*sizeof(uint), hipMemcpyDeviceToHost));
cout << "Total time is " << Timer.ElapsedMillis() <<endl;   

/*    for (int i = 0; i < str_length+1; ++i)
    {
        for (int j = keys_sa[i]-1; j < str_length+1; ++j)
        {
            cout << str[j];
        }
        cout << keys_sa[i] <<endl;
    }
*/  
//myfile.close();
   printf("================ SA completed ====================\n");

    _SafeDeleteArray(keys_sa);
    _SafeDeleteArray(str);
    _SafeDeleteArray(str_value);
    CUDA_SAFE_CALL(hipFree(d_keys_sa));
    CUDA_SAFE_CALL(hipFree(d_str));
    return 0;
}
}
